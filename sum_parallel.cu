#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void sumArray(float *array, float *result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        atomicAdd(result, array[idx]);
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <array_size>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    if (N <= 0) {
        printf("Array size must be positive\n");
        return 1;
    }

    float *h_array = (float *)malloc(N * sizeof(float));
    float *d_array, *d_result;
    float h_result = 0.0f;
    
    // Инициализация массива
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_array[i] = (float)rand() / RAND_MAX;
    }

    // Выделение памяти на устройстве
    hipMalloc(&d_array, N * sizeof(float));
    hipMalloc(&d_result, sizeof(float));
    hipMemcpy(d_array, h_array, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    // Запуск ядра
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    sumArray<<<numBlocks, blockSize>>>(d_array, d_result, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Копирование результата обратно
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    printf("Parallel sum: %f\n", h_result);
    printf("Time: %f seconds\n", milliseconds / 1000);

    // Освобождение памяти
    hipFree(d_array);
    hipFree(d_result);
    free(h_array);

    return 0;
}
