#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void arrayOperations(float *A, float *B, float *add, float *sub, float *mul, float *div, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        add[idx] = A[idx] + B[idx];
        sub[idx] = A[idx] - B[idx];
        mul[idx] = A[idx] * B[idx];
        div[idx] = A[idx] / B[idx];
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <array_size>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    if (N <= 0) {
        printf("Array size must be positive\n");
        return 1;
    }

    float *h_A = (float *)malloc(N * sizeof(float));
    float *h_B = (float *)malloc(N * sizeof(float));
    float *h_add = (float *)malloc(N * sizeof(float));
    float *h_sub = (float *)malloc(N * sizeof(float));
    float *h_mul = (float *)malloc(N * sizeof(float));
    float *h_div = (float *)malloc(N * sizeof(float));
    
    float *d_A, *d_B, *d_add, *d_sub, *d_mul, *d_div;

    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        h_A[i] = (float)rand() / RAND_MAX;
        h_B[i] = (float)rand() / RAND_MAX + 0.1f;
    }

    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_add, N * sizeof(float));
    hipMalloc(&d_sub, N * sizeof(float));
    hipMalloc(&d_mul, N * sizeof(float));
    hipMalloc(&d_div, N * sizeof(float));

    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    arrayOperations<<<numBlocks, blockSize>>>(d_A, d_B, d_add, d_sub, d_mul, d_div, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_add, d_add, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_sub, d_sub, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_mul, d_mul, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_div, d_div, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Parallel array operations completed\n");
    printf("Time: %f seconds\n", milliseconds / 1000);

    // Пример вывода первых 5 элементов
    printf("Sample results (first 5 elements):\n");
    for (int i = 0; i < 5 && i < N; i++) {
        printf("%f + %f = %f\n", h_A[i], h_B[i], h_add[i]);
        printf("%f - %f = %f\n", h_A[i], h_B[i], h_sub[i]);
        printf("%f * %f = %f\n", h_A[i], h_B[i], h_mul[i]);
        printf("%f / %f = %f\n", h_A[i], h_B[i], h_div[i]);
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_add); hipFree(d_sub); hipFree(d_mul); hipFree(d_div);
    free(h_A); free(h_B); free(h_add); free(h_sub); free(h_mul); free(h_div);

    return 0;
}
