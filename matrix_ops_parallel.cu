#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ROWS 1000
#define COLS 1000
#define N (ROWS * COLS)

__global__ void matrixOpsKernel(float *a, float *b, float *add, float *sub, 
                               float *mul, float *div, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        add[idx] = a[idx] + b[idx];
        sub[idx] = a[idx] - b[idx];
        mul[idx] = a[idx] * b[idx];
        div[idx] = a[idx] / (b[idx] + 0.0001f);
    }
}

int main(int argc, char **argv) {
    if (argc != 3) {
        return 1;
    }
    
    int blockDimX = atoi(argv[1]);
    int blockDimY = atoi(argv[2]);
    
    if (blockDimX <= 0 || blockDimY <= 0 || blockDimX * blockDimY > 1024) {
        return 1;
    }

    
    float *h_a = (float*)malloc(N * sizeof(float));
    float *h_b = (float*)malloc(N * sizeof(float));
    float *h_add = (float*)malloc(N * sizeof(float));
    float *h_sub = (float*)malloc(N * sizeof(float));
    float *h_mul = (float*)malloc(N * sizeof(float));
    float *h_div = (float*)malloc(N * sizeof(float));
    
    float *d_a, *d_b, *d_add, *d_sub, *d_mul, *d_div;
    
    for (int i = 0; i < N; i++) {
        h_a[i] = (float)rand() / RAND_MAX;
        h_b[i] = (float)rand() / RAND_MAX;
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_add, N * sizeof(float));
    hipMalloc(&d_sub, N * sizeof(float));
    hipMalloc(&d_mul, N * sizeof(float));
    hipMalloc(&d_div, N * sizeof(float));
    
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 block(blockDimX, blockDimY);
    dim3 grid((COLS + block.x - 1) / block.x, (ROWS + block.y - 1) / block.y);
    
    hipEventRecord(start);
    
    matrixOpsKernel<<<grid, block>>>(d_a, d_b, d_add, d_sub, d_mul, d_div, ROWS, COLS);
    
    hipMemcpy(h_add, d_add, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_sub, d_sub, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_mul, d_mul, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_div, d_div, N * sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("Размер матрицы: %dx%d\n", ROWS, COLS);
    printf("Время: %f seconds\n\n", milliseconds / 1000.0f);
    
    hipFree(d_a); hipFree(d_b); hipFree(d_add); 
    hipFree(d_sub); hipFree(d_mul); hipFree(d_div);
    free(h_a); free(h_b); free(h_add); free(h_sub); free(h_mul); free(h_div);
    
    return 0;
}
