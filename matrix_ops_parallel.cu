#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ROWS 1000
#define COLS 1000
#define N (ROWS * COLS)
#define BLOCK_SIZE 16

__global__ void matrixOpsKernel(float *a, float *b, float *add, float *sub, float *mul, float *div, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        add[idx] = a[idx] + b[idx];
        sub[idx] = a[idx] - b[idx];
        mul[idx] = a[idx] * b[idx];
        div[idx] = a[idx] / b[idx];
    }
}

int main() {
    float *h_a = (float*)malloc(N * sizeof(float));
    float *h_b = (float*)malloc(N * sizeof(float));
    float *h_add = (float*)malloc(N * sizeof(float));
    float *h_sub = (float*)malloc(N * sizeof(float));
    float *h_mul = (float*)malloc(N * sizeof(float));
    float *h_div = (float*)malloc(N * sizeof(float));
    
    float *d_a, *d_b, *d_add, *d_sub, *d_mul, *d_div;
    
    // Инициализация матриц
    for (int i = 0; i < N; i++) {
        h_a[i] = (float)rand() / RAND_MAX + 0.1f;
        h_b[i] = (float)rand() / RAND_MAX + 0.1f;
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_add, N * sizeof(float));
    hipMalloc(&d_sub, N * sizeof(float));
    hipMalloc(&d_mul, N * sizeof(float));
    hipMalloc(&d_div, N * sizeof(float));
    
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((COLS + block.x - 1) / block.x, (ROWS + block.y - 1) / block.y);
    
    hipEventRecord(start);
    
    matrixOpsKernel<<<grid, block>>>(d_a, d_b, d_add, d_sub, d_mul, d_div, ROWS, COLS);
    
    hipMemcpy(h_add, d_add, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_sub, d_sub, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_mul, d_mul, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_div, d_div, N * sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("First add result: %f\n", h_add[0]);
    printf("First sub result: %f\n", h_sub[0]);
    printf("First mul result: %f\n", h_mul[0]);
    printf("First div result: %f\n", h_div[0]);
    printf("Time: %f seconds\n", milliseconds / 1000.0f);
    
    hipFree(d_a); hipFree(d_b); hipFree(d_add); hipFree(d_sub); hipFree(d_mul); hipFree(d_div);
    free(h_a); free(h_b); free(h_add); free(h_sub); free(h_mul); free(h_div);
    
    return 0;
}
