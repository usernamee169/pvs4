#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N1 131072   // 2^17
#define N2 524288   // 2^19
#define N3 1048576  // 2^20

#define THREADS 256

__device__ void swap(float &a, float &b) {
    float t = a;
    a = b;
    b = t;
}

__global__ void bitonicSortStep(float *devValues, int j, int k) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;
    
    if (ixj > i) {
        if ((i & k) == 0) {
            if (devValues[i] > devValues[ixj]) {
                swap(devValues[i], devValues[ixj]);
            }
        }
        if ((i & k) != 0) {
            if (devValues[i] < devValues[ixj]) {
                swap(devValues[i], devValues[ixj]);
            }
        }
    }
}

void testSort(int size) {
    float *h_values = (float*)malloc(size * sizeof(float));
    float *d_values;
    
    for (int i = 0; i < size; i++) {
        h_values[i] = (float)rand() / RAND_MAX;
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc(&d_values, size * sizeof(float));
    hipMemcpy(d_values, h_values, size * sizeof(float), hipMemcpyHostToDevice);
    
    int blocks = (size + THREADS - 1) / THREADS;
    
    hipEventRecord(start);
    
    int j, k;
    for (k = 2; k <= size; k <<= 1) {
        for (j = k >> 1; j > 0; j >>= 1) {
            bitonicSortStep<<<blocks, THREADS>>>(d_values, j, k);
        }
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(h_values, d_values, size * sizeof(float), hipMemcpyDeviceToHost);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    int sorted = 1;
    for (int i = 0; i < size - 1; i++) {
        if (h_values[i] > h_values[i+1]) {
            sorted = 0;
            break;
        }
    }
    
    printf("Размер массива: %d\n", size);
    printf("Время: %f seconds\n\n", milliseconds / 1000.0f);
    
    hipFree(d_values);
    free(h_values);
}

int main() {
    srand(time(NULL));
    
    printf("Параллельная сортировка:\n");
    testSort(N1);
    testSort(N2);
    testSort(N3);
    
    return 0;
}
